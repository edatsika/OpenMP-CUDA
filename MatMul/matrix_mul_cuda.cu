#include <iostream>
#include <hip/hip_runtime.h>

#define SIZE 1024

__global__ void matrixMultiplyCUDA(float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < SIZE && col < SIZE) {
        float sum = 0.0f;
        for (int k = 0; k < SIZE; ++k)
            sum += A[row * SIZE + k] * B[k * SIZE + col];
        C[row * SIZE + col] = sum;
    }
}

int main() {
    int N = SIZE * SIZE;
    size_t bytes = N * sizeof(float);

    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    for (int i = 0; i < N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks(SIZE / threads.x, SIZE / threads.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matrixMultiplyCUDA<<<blocks, threads>>>(d_A, d_B, d_C);
    hipEventRecord(stop);

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "CUDA Time: " << milliseconds / 1000.0 << " seconds\n";

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}